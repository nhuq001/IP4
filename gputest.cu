
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef struct species //holds diffusion and reaction rates for a species
{
    int diffusion_rate;
    int reaction_rate;
}species;

__device__ unsigned int RNG()
{
	unsigned int m_w = 150 + threadIdx.x;
    unsigned int m_z = 40 + (threadIdx.x / 2);

    for(int i=0; i < 100; i++)
    {
        m_z = 36969 * (m_z & 65535) + (m_z >> 16);
        m_w = 18000 * (m_w & 65535) + (m_w >> 16);
	}
	return (m_z << 16) + m_w;
}

__device__ int sum_reactions(int element, int *config, struct species *all)
{
	int total_reactions = 0;
	
	//find row of configuration matrix by element / (columns in rate matrix)
	//rate matrix always has 3 columns
	int row = element / 3;
	
	//iterate through the columns of config and update total reactions.
	int i;
	for (i = 0; i < 2; i++) //only 2 columns for demo
		total_reactions = total_reactions + (config[row * 2 + i] * all[i].reaction_rate); //add last result to current result;
	
	return total_reactions;
}

__device__ int sum_diffusions(int element, int *config, struct species *all)
{
	int total_diffusion = 0;
	
	//find row of configuration matrix by element / (columns in rate matrix)
	//rate matrix always has 3 columns
	int row = element / 3;
	
	//iterate through the columns of config and update total diffusions.
	int i;
	for (i = 0; i < 2; i++) //only 2 columns for demo
		total_diffusion = total_diffusion + (config[row * 2 + i] * all[i].diffusion_rate); //add last result to current result;
	
	return total_diffusion;
}

__global__ void rate_matrix_1(int *config, int *rate, struct species *all)//species is a struct that holds reaction and diffusion rates of a species
{
	printf("Config [0] %i\n", config[0]);
	//element will be used for rate[index]
	int element = blockIdx.x*blockDim.x + threadIdx.x; //index for this instance based on which core and thread is running
													   // blockDim.x is the total amount of threads in a core
	if((element + 1) % 3 == 1) //first column
		rate[element] = sum_reactions(element, config, all);
	if((element + 1) % 3 == 2) //second column
		rate[element] = sum_diffusions(element, config, all);
}

__global__ void rate_matrix_2(int *rate)
{
	//element will be used for rate[index]
	int element = blockIdx.x*3+2; //get 3rd column of rate matrix
	rate[element] = rate[element - 1] + rate[element - 2]; //sums reaction and diffusion rate for subvolume x.
}

__device__ void diffusion(int *conn_matrix, int *con_matrix, int element)
{
    //find random element in the conn_matrix
	int conn_element = element * 6;//gets row in connectivity matrix
    int r1 = RNG() % 6;
    int sv2 = conn_matrix[conn_element + r1]; //diffusion target
    int r2 = RNG() % 2; //gives random column in configuration matrix
	
	int r3;
	if(con_matrix[(element * 2) + r2] != 0) //avoid divide by 0
		r3 = RNG() % con_matrix[(element * 2) + r2]; //take random amount of particle from sv1
	else
		r3 = 0;
	
	int r4;
	if(con_matrix[(sv2 * 2) + r2] != 0) //avoid divide by 0
		r4 = RNG() % con_matrix[(sv2 * 2) + r2]; //take random amount of particle from sv2
	else
		r4 = 0;
	
    con_matrix[(element * 2) + r2] = con_matrix[(element * 2) + r2] + r4 - r3; //change amount in sv1
    con_matrix[(sv2 * 2) + r2] = con_matrix[(sv2 * 2) + r2] + r3 - r4; //change amount in sv2
}

__device__ void reaction(int *con_matrix, int element)
{
	int r1 = RNG() % 2;//get a random number to decide which reaction occurs
	if(r1 == 0 && con_matrix[element * 2] > 0) //a turns to b
	{
		con_matrix[element * 2] = con_matrix[element * 2] - 1;
		con_matrix[(element * 2) + 1] = con_matrix[(element * 2) + 1] + 1;
	}
	else if(r1 == 1 && con_matrix[(element * 2) + 1] > 0)
	{
		con_matrix[element * 2] = con_matrix[element * 2] + 1;
		con_matrix[(element * 2 ) + 1] = con_matrix[(element * 2 )+ 1] - 1;
	}
}

__global__ void NSM (int *conn_Matrix, int *rate_Matrix, int *con_matrix, struct species *all)
{
  int element = blockIdx.x * blockDim.x + threadIdx.x; //assigns subvolume to thread.
  int r1 = RNG() % 3; //will diffuse, react, or neither
  if(r1 == 0)
      diffusion(conn_Matrix, con_matrix, element);
  else if (r1 == 1)
      reaction(con_matrix, element);
  //update rate matrix
  rate_Matrix[element * 3] = con_matrix[element * 2] * all[0].diffusion_rate + con_matrix[(element * 2) + 1] * all[1].diffusion_rate;
  rate_Matrix[(element * 3) + 1] = con_matrix[element * 2] * all[0]. reaction_rate + con_matrix[(element * 2) + 1] * all[1].reaction_rate;
  rate_Matrix[(element * 3) + 2] = rate_Matrix[(element * 3) + 1] + rate_Matrix[element * 3];
}

void duplicate_connectivity_matrix(int dupes, int *original, int *clone) //make disconnected geometry that is identical to the first
{
    int i, j, k; //necessary for loop index
    //first nested loop makes a copy of the first
    for (i = 0; i < 8; i++)
        for (j = 0; j < 6; j++)
            clone[i * 6 + j] = original[i * 6 + j];
    //second nested loops clones the geometry but is unconnected to the previous
    for(i = 0; i < dupes; i++)
        for(j = 0; j < dupes*8; j++)
            for(k = 0; k < 6; k++)
                clone[(j + 8) * 6 + k ] = clone[j * 6 + k] + 8;
}

void populate_subvolumes (int size, int *config)//pass total amount in array and the array
{
    int i;
    for (i = 0 ; i < size ; i++)
    {
        config[i] = rand()%20;
    }
}


int main()
{
    srand(time(NULL)); //needed for random value
    int sv = 1;
    int con_matrix1 [8 * 6] = {			   //premade geometry for connectivity matrix
                              1,0,2,0,4,0, //1
                              1,0,3,1,5,1, //2
                              3,2,2,0,6,2, //3
                              3,2,3,1,7,3, //4
                              5,4,6,4,4,0, //5
                              5,4,7,5,5,1, //6
                              7,6,6,4,6,2, //7
                              7,6,7,5,7,3  //8
                              };

    int con_matrix2 [(sv + 1)* 8 * 6];
    duplicate_connectivity_matrix(sv, con_matrix1, con_matrix2);
	
	printf("Connectivity Matrix:\n");
	int k;
    int j;
    for(k = 0; k < sv*8 + 8; k++)
    {
        printf("%d [", k);
         for(j = 0; j < 5; j++)
              printf("%d, ",con_matrix2[k*6 + j]);
         printf("%d]\n", con_matrix2[k*6 + 5]);
    }
	printf("\n");
	
	int config_matrix[(sv+1) * 8 * 2]; //only 2 species in model
	populate_subvolumes((sv+1) * 8 * 2, config_matrix);
	
	printf("Configuration Matrix:\n");
	for(k = 0; k < sv*8 + 8; k++)
    {
        printf("%d [%d, %d]\n", k, config_matrix[k*2], config_matrix[(k*2)+1]);
    }
	printf("\n");
	
	struct species types[2];
	types[0].diffusion_rate = 1;	types[0].reaction_rate = 1;
	types[1].diffusion_rate = 2;	types[1].reaction_rate = 2;
	
	int rate_matrix[(sv+1) * 8 * 3];//create rate matrix
	//parallelization starts here
	
	printf("Config Outside[0] %d\n", config_matrix[0]);
    //rate_matrix_1
	int *gpu_A;
	int *gpu_B;
	species *gpu_C;
	hipMalloc(&gpu_A, (sv + 1)*8*2*sizeof(int));
	hipMemcpy(gpu_A, &config_matrix, (sv + 1)*8*2*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&gpu_B, (sv + 1)*8*3*sizeof(int));
	hipMalloc(&gpu_C, 2*sizeof(species));
	hipMemcpy(gpu_C, &types, 2*sizeof(species), hipMemcpyHostToDevice);
	rate_matrix_1<<<1, 1>>>(config_matrix, rate_matrix, types);
	hipMemcpy(&rate_matrix, gpu_C, (sv + 1)*8*3*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpu_A);
	hipFree(gpu_B);
	hipFree(gpu_C);
	
	
	/*rate_matrix_2
	cudaMalloc(&gpuA, N*sizeof(float)); // Allocate enough memory on the GPU
    cudaMemcpy(gpuA, a, N*sizeof(float), cudaMemcpyHostToDevice); // Copy array from CPU to GPU
    rate_matrix_2<<<numCores, numThreads>>>(rate_matrix);  // Call GPU Sqrt
    cudaMemcpy(a, gpuA, N*sizeof(float), cudaMemcpyDeviceToHost); // Copy array from GPU to CPU
    cudaFree(&gpuA); // Free the memory on the GPU
	  rate_matrix2*/
	  
	/*printf("Initial Rate Matrix: \n");
	for(k = 0; k < sv*8 + 8; k++)
    {
        printf("%d [", k);
         for(j = 0; j < 2; j++)
              printf("%d, ",rate_matrix[k*2 + j]);
         printf("%d]\n", con_matrix2[k*2 + 2]);
    }*/
	
	
	/*NSM
	cudaMalloc((void**)&gpuA, N*sizeof(float)); // Allocate enough memory on the GPU
    (gpuA, a, N*sizeof(float), cudaMemcpyHostToDevice); // Copy array from CPU to GPU
    rate_matrix_2<<<numCores, numThreads>>>(rate_matrix);  // Call GPU Sqrt
    cudaMemcpy(a, gpuA, N*sizeof(float), cudaMemcpyDeviceToHost); // Copy array from GPU to CPU
    cudaFree(&gpuA); // Free the memory on the GPU
	  NSM*/
	
	//parallelization ends here
	
	printf("Altered Configuration Matrix:\n");
	for(k = 0; k < sv*8 + 8; k++)
    {
        printf("%d [%d, %d]\n", k, config_matrix[k*2], config_matrix[(k*2)+1]);
    }
	
    return 0;
}